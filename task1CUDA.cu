#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// ���� ��� ������������ ������� (1 ����, 256 �������)
__global__ void cuda_array_sum(long long* result, int* array, int size) {
    __shared__ int shared_sum[256]; // Shared memory ��� ������������� ����

    int tid = threadIdx.x;
    int index = tid;
    int stride = blockDim.x;

    // ������ ����� ��������� ���� ��������
    long long local_sum = 0;
    for (int i = index; i < size; i += stride) {
        local_sum += array[i];
    }
    shared_sum[tid] = local_sum;
    __syncthreads();

    // �������� � shared memory (��������� ���������� �������)
    for (int s = 128; s > 0; s >>= 1) {
        if (tid < s) {
            shared_sum[tid] += shared_sum[tid + s];
        }
        __syncthreads();
    }

    // ������ ����� ���������� ���������
    if (tid == 0) {
        *result = shared_sum[0];
    }
}

int main(int argc, char* argv[]) {
    int array_size = 0;
    int* array = NULL;
    int* cuda_array = NULL;
    long long sum = 0;
    long long* cuda_sum = NULL;

    if (argc != 2) {
        printf("Usage: %s <array_size>\n", argv[0]);
        return 1;
    }

    array_size = atoi(argv[1]);
    if (array_size <= 10000) {
        fprintf(stderr, "Error: Array size must be greater than 10000\n");
        return 1;
    }

    array = (int*)malloc(array_size * sizeof(int));
    if (array == NULL) {
        fprintf(stderr, "Error: Memory allocation failed\n");
        return 1;
    }

    srand(time(NULL));
    for (int i = 0; i < array_size; i++) {
        array[i] = rand() % 1000;
    }

    clock_t start_time = clock();

    // ��������� ������ GPU
    hipMalloc((void**)&cuda_array, sizeof(int) * array_size);
    hipMalloc((void**)&cuda_sum, sizeof(long long));

    // ������������� ����� �� ����������
    long long zero = 0;
    hipMemcpy(cuda_sum, &zero, sizeof(long long), hipMemcpyHostToDevice);

    // ����������� ������ � GPU
    hipMemcpy(cuda_array, array, sizeof(int) * array_size, hipMemcpyHostToDevice);

    // ���������� ����������
    cuda_array_sum << <1, 256 >> > (cuda_sum, cuda_array, array_size);

    // ����������� ����������
    hipMemcpy(&sum, cuda_sum, sizeof(long long), hipMemcpyDeviceToHost);

    // ������� ������ GPU
    hipFree(cuda_array);
    hipFree(cuda_sum);

    clock_t end_time = clock();
    double elapsed_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;

    free(array);

    FILE* f = fopen("cuda_sum_time.txt", "a");
    if (f == NULL) {
        fprintf(stderr, "Error: Cannot open output file\n");
        return 1;
    }
    fprintf(f, "%lf\n", elapsed_time);
    fclose(f);

    return 0;
}
